#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include ""
#include <hip/device_functions.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <time.h>
#include "hip/hip_runtime_api.h"

__global__ void CMM_CUDA_kernel(const int *dim, int *m, int *result, int n)
{
	int number_of_threads = n, i, min, j, k, index;
	index = threadIdx.x;
	i = index + 1;
	j = index + 1;
	m[(n + 1) * i + i] = 0;
	__syncthreads();

	while (number_of_threads >= 1){
		number_of_threads = number_of_threads - 1;
		if (index < number_of_threads){
			j = j + 1;
			min = INT_MAX;
			for (k = i; k <= j - 1; k++)
			if (m[(n + 1) * i + k] + m[(n + 1) * (k + 1) + j] + dim[i - 1] * dim[k] * dim[j] < min)
				min = m[(n + 1) * i + k] + m[(n + 1) * (k + 1) + j] + dim[i - 1] * dim[k] * dim[j];
			m[(n + 1) * i + j] = min;
		}
		__syncthreads();
	}
	if (index == 0) result[0] = m[(n + 1) * 1 + n];
}

__global__ void CMM_CUDA_with_Data_Layout_kernel(const int *dim, int *m, int *result, int n)
{
	int number_of_threads = n, i, min, j, k, index; int xprim, yprim, d;
	index = threadIdx.x;
	i = index + 1;
	j = index + 1;
	m[n + 1 + i] = 0;         //---- m[i,i]

	__syncthreads();

	while (number_of_threads >= 1){
		number_of_threads = number_of_threads - 1;
		if (index < number_of_threads){
			j = j + 1;
			min = INT_MAX;
			for (k = i; k <= j - 1; k++){
				d = k - i;  xprim = n + 1 + d*n - (d*(d - 1) / 2) + i;                      //---- m[i,k]
				d = j - k - 1;  yprim = n + 1 + d*n - (d*(d - 1) / 2) + k + 1;    //---- m[k+1,j]
				if (m[xprim] + m[yprim] + dim[i - 1] * dim[k] * dim[j] < min)
					min = m[xprim] + m[yprim] + dim[i - 1] * dim[k] * dim[j];
			}
			d = j - i;  xprim = n + 1 + d*n - (d*(d - 1) / 2) + i;  //---- m[i,j]
			m[xprim] = min;
			//printf("m[%d]=%d\n", xprim,m[xprim]);
		}
		__syncthreads();
	}
	if (index == 0) {
		xprim = (n + 1) * (n + 2) / 2;  //---- m[1,n]
		result[0] = m[xprim];
	}
}

int CMM_Serial(int*, int, float*);
int CMM_CUDA(int*, int, float*);
int CMM_CUDA_with_Data_Layout(int*, int, float*);

int main()
{
	FILE *out = fopen("output.txt", "w");
	int i, n, *dim;	hipError_t cudaStatus;
	float time1 = 0, time2 = 0, time3 = 0;
	int result1 = 0, result2 = 0, result3 = 0;

	for (n = 10; n <= 1024; n++){

		dim = new int[n + 1];
		for (i = 0; i <= n; i++) dim[i] = (int)rand() / (RAND_MAX + 1) * (10 - 2) + 2;

		result1 = CMM_Serial(dim, n, &time1);
		result2 = CMM_CUDA(dim, n, &time2);
		result3 = CMM_CUDA_with_Data_Layout(dim, n, &time3);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess){ printf("for n=%d , hipDeviceReset failed!", n);	break; }

		if (result1 != result2 || result1 != result3){ printf("for n=%d , results are wrong!\n", n); break; }

		printf("n = %d   ,time1 = %f   ,time2 = %f  ,time3 = %f \n", n, time1, time2, time3);
		fprintf(out, "%d %f %f %f\n", n, time1, time2, time3);


		delete[] dim;

	}
	fclose(out);
	return 0;
}

int CMM_Serial(int* dim, int n, float *time)
{
	float elapsed_time;
	hipEvent_t start, stop; hipEventCreate(&start); hipEventCreate(&stop);

	int i, j, k, diagonal, min, temp;
	int** m = new int*[n + 1];
	for (i = 0; i <= n; i++) m[i] = new int[n + 1];

	hipEventRecord(start, 0);

	for (i = 0; i <= n; i++) m[i][i] = 0;
	for (diagonal = 1; diagonal < n; diagonal++){
		for (i = 1; i <= n - diagonal; i++){
			j = i + diagonal;
			min = INT_MAX;
			for (k = i; k < j; k++){
				temp = m[i][k] + m[k + 1][j] + dim[i - 1] * dim[k] * dim[j];
				if (temp < min) min = temp;
			}
			m[i][j] = min;
		}
	}
	temp = m[1][n];

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time, start, stop);
	*time = elapsed_time;


	delete[] m;

	return temp;
}

int CMM_CUDA(int* dim, int n, float *time)
{
	float elapsed_time;
	hipEvent_t start, stop; hipEventCreate(&start); hipEventCreate(&stop);

	int i, *dev_m, *dev_dim, *dev_result, result[1];

	hipMalloc((void**)&dev_result, 1 * sizeof(int));
	hipMalloc((void**)&dev_dim, (n + 1) * sizeof(int));
	hipMalloc((void**)&dev_m, (n + 1) * (n + 1) * sizeof(int));
	hipMemcpy(dev_dim, dim, (n + 1) * sizeof(int), hipMemcpyHostToDevice);

	hipEventRecord(start, 0);

	for (i = 1; i <= 10; i++) CMM_CUDA_kernel << <1, n >> >(dev_dim, dev_m, dev_result, n);
	hipDeviceSynchronize();

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time, start, stop);
	*time = elapsed_time / 10.0;


	hipMemcpy(result, dev_result, 1 * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(dev_result); hipFree(dev_dim); hipFree(dev_m);

	return result[0];
}

int CMM_CUDA_with_Data_Layout(int* dim, int n, float *time)
{
	float elapsed_time;
	hipEvent_t start, stop; hipEventCreate(&start); hipEventCreate(&stop);

	int i, *dev_m, *dev_dim, *dev_result, result[1];

	hipMalloc((void**)&dev_result, 1 * sizeof(int));
	hipMalloc((void**)&dev_dim, (n + 1) * sizeof(int));
	hipMalloc((void**)&dev_m, (n + 1) * (n + 1) * sizeof(int));
	hipMemcpy(dev_dim, dim, (n + 1) * sizeof(int), hipMemcpyHostToDevice);

	hipEventRecord(start, 0);

	for (i = 1; i <= 10; i++) CMM_CUDA_with_Data_Layout_kernel << <1, n >> >(dev_dim, dev_m, dev_result, n);
	hipDeviceSynchronize();

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time, start, stop);
	*time = elapsed_time / 10.0;

	hipMemcpy(result, dev_result, 1 * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(dev_result); hipFree(dev_dim); hipFree(dev_m);

	return result[0];
}
